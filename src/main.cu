#include "hip/hip_runtime.h"
#include "include.h"

#include "constants.h"

#include <hip/hip_runtime.h>
#include <>

#include <chrono>
#include <iomanip>
#include <fstream>
#include <sstream>

using std::cout, std::cerr, std::endl;
using std::vector, std::string, std::make_shared, std::shared_ptr;
using std::stoi, std::stoul, std::min, std::max, std::numeric_limits, std::abs;

int g_worldLogIdx = -1;
string g_worldLogOutDir = "";
std::ofstream g_worldLogStream;

static const size_t g_numParticles = NUM_PARTICLES;
static size_t g_numWorlds, g_maxBlocks;
static int* h_convergenceFlags;
static bool g_isGlobalConverged(false);
std::chrono::high_resolution_clock::time_point g_progStart;
float g_curStepTime(0.0f);
long long g_curStep(0);

// Device Hyperparameters - Constant Space //
__constant__ size_t d_numParticles;
__constant__ size_t d_numWorlds, d_numPlanes;
__constant__ glm::vec4 d_planeP[6], d_planeN[6];

bool g_is_simFrozen(false);
hipEvent_t cudaEvt_simStart, cudaEvt_simStop;

// blocks = k, threads = n
dim3 g_blocksPerGrid;
dim3 g_threadsPerBlock;

// static const int g_timeSampleSz = KERNEL_TIMING_SAMPLESZ;
static size_t g_timeSampleCt = 0;
static float g_totalKernelTime(0.0f), g_totalBatchLoopTime(0.0f);

ParticleData g_particles;
PlaneData g_planes;

static void init() {
    srand(0);
    size_t k = g_numWorlds;

    // CUDA //
        gpuErrchk(hipSetDevice(0));
        hipEventCreate(&cudaEvt_simStart);
        hipEventCreate(&cudaEvt_simStop);

    // Planes //
        const float plane_width = 540.0f;
        g_planes = PlaneData(6, plane_width);
        g_planes.initPlanes();
        g_planes.copyToDevice();

    // Particles //
        g_particles = ParticleData(g_numParticles, k);
        g_particles.init(0.5f);
        g_particles.copyToDevice();

        // numWorlds
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_numWorlds), &g_numWorlds, sizeof(size_t)));

        // We should zero h_convergenceFlags here and send that to CUDA
        h_convergenceFlags = new int[k];
        memset(h_convergenceFlags, 0, k * sizeof(int));
        hipMemcpy(g_particles.d_convergenceFlags, h_convergenceFlags, k * sizeof(int), hipMemcpyHostToDevice);
}

/*
    Instead of iterating over each particle, we will make a kernel that runs for each particle
*/

// Assume mass is 1; F / 1 = A
__device__ glm::vec3 getAcceleration(int idx, const glm::vec4* v) {
    float mass = 1.0f;

    // Simple force composed of gravity and air resistance
    glm::vec3 F_total = glm::vec3(0.0f, GRAVITY, 0.0f) - ((AIR_FRICTION / mass) * glm::vec3(v[idx])); 

    return F_total;
}

/*
    In a flocking simulation, you might have various rules and applications to follow - in this case, we are just
    going to establish a simple distance constraint that is resolve with impulse / momentum.
*/

static __device__ void solveConstraints(int idx, const glm::vec4* pos, const glm::vec4* vel, const float* radii,
                                 glm::vec3& x_new, glm::vec3& v_new, float& dt, const glm::vec3& a,
                                 int simulationIdx, int particleIdx, glm::vec3* s_dv, int& is_converged) {
    // Truncate the w component
    glm::vec3 x(pos[idx]), v(vel[idx]);
    const float r_i = radii[idx];

    // Particle-Particle Collisions //
    /*
        This could be below plane collisions, but seeing as we synchronized threads, we will do it here

        This is the inner loop of for i in range(particles), for j in range(i + 1, particles)

        We can grab the global array value as simulationIdx * particles + j. Note that because we
        are handling j > i particles in the ith thread, the jth thread will never see i - so we should
        update the opposite of the impulse from i -> j to the jth shared velocity; 
    */
    for (int j = particleIdx + 1; j < d_numParticles; j++) {
        int idx_j = simulationIdx * d_numParticles + j;

        glm::vec3 x_j(pos[idx_j]), v_j(vel[idx_j]);
        float r_j = radii[idx_j];

        /*
            If the distance from our particle to the other is less than radii[i] + radii[j] we have collided.

            We can take the direction of j to x and say we (particle i) should be pushed in that direction.

            The extent to which we move, or impulse, is dependent on the relative velocity, or how fast we
            are moving towards each other. For example, if v_rel < 0, we are moving towards each other, and we
            should push off more.

            J = [(1 + e) * v_rel] / [1/m1 + 1/m2]
        */
        glm::vec3 x_ij = x - x_j;
        float d_ij = glm::length(x_ij);

        if (d_ij < r_i + r_j) {
            glm::vec3 n_ij = glm::normalize(x_ij);
            glm::vec3 v_ij = v - v_j;

            float v_rel = glm::dot(v_ij, n_ij);
            float impulse = (1 + RESTITUTION) * v_rel / (1 + 1);

            glm::vec3 impulse_vec = impulse * n_ij;
            s_dv[particleIdx] += impulse_vec;

            // Consequently, we should change the velocities of j. This is not thread safe so we have to atomic
            atomicAdd(&s_dv[j].x, -impulse_vec.x);
            atomicAdd(&s_dv[j].y, -impulse_vec.y);
            atomicAdd(&s_dv[j].z, -impulse_vec.z);
        }
    }

    // Synchronize threads, because we don't want to start plane collisions until this is done
    __syncthreads();
    v += s_dv[particleIdx];
    
    // Plane Collisions //
    for (int i = 0; i < d_numPlanes; i++) {
        const glm::vec3 p(d_planeP[i]), n(d_planeN[i]);

        glm::vec3 new_p = p + (r_i * n);

        float d_0 = glm::dot(x - new_p, n);
        float d_n = glm::dot(x_new - new_p, n);

        glm::vec3 v_tan = v - (glm::dot(v, n) * n);
        v_tan = (1 - FRICTION) * v_tan;

        if (d_n < FLOAT_EPS) {
            float f = d_0 / (d_0 - d_n);
            dt = f * dt;

            glm::vec3 v_collision = (v + (dt * a)) * RESTITUTION;    
            glm::vec3 x_collision = x;

            x_new = x_collision;
            v_new = (abs(glm::dot(v_collision, n)) * n) + (v_tan);
        }

        // Convergence or jitter check - (v = 0, "on" the plane, and acceleration towards plane)
        if ((length(v_new) < STOP_VELOCITY) && (d_n < STOP_PLANE_DIST) && (dot(a, n) < FLOAT_EPS)) {
            v_new = glm::vec4(0.0f);
            is_converged = 1;
        } 
    }
}

/*
    We are now working with global arrays of great size; 
*/
__global__ void simulateKernel(glm::vec4* pos, glm::vec4* vel, float* radii, int* convergeFlags) {
    unsigned int simulationIdx(blockIdx.x), particleIdx(threadIdx.x);

    // Overflow shouldn't be possible
    int idx = simulationIdx * d_numParticles + particleIdx;

    // FIXME: When a thread returns early, it cannot join __syncthreads later, so we should note this
    if (idx >= (d_numWorlds * d_numParticles)) {
        printf("Returning idx = %d\n", idx);        
        return;
    }

    // Allocate shared memory for graceful impulse & convergence handling; each block is one world so this works 
    __shared__ glm::vec3 s_dv[NUM_PARTICLES];
    __shared__ int s_converged;

    // We only want to initialize it once
    if (particleIdx == 0) {
        s_converged = 1;
    }

    // Handle fractional timesteps
    float dt_remaining = DT_SIMULATION;
    float dt = dt_remaining;
    short max_iter = 10;
    int is_stopped = 0;
    
    glm::vec3 x_cur(pos[idx]), v_cur(vel[idx]);
    glm::vec3 x_new(x_cur), v_new(v_cur);

    while (max_iter && dt_remaining > FLOAT_EPS) {
        // Within the timestep multiple collisions are possible, so we will have to reuse the shared memory 
        s_dv[particleIdx] = glm::vec3(0.0f);
        
        glm::vec3 a = getAcceleration(idx, vel);

        // Integrate over timestep to update
        x_new = x_cur + (dt * v_cur);
        v_new = v_cur + (dt * a);

        // We have to synchronize before and after entering
        __syncthreads();

        // Resolve particle-particle AND particle-plane position constraints
        solveConstraints(idx, pos, vel, radii, x_new, v_new, dt, a, simulationIdx, particleIdx, s_dv, is_stopped);

        __syncthreads();

        x_cur = x_new;
        v_cur = v_new;

        dt_remaining -= dt;
        max_iter--;
    }

    // The and of all 64 particles being stopped in this world being 1 represents full convergence
    atomicAnd(&s_converged, is_stopped);

    // No need for atomic here, only the first thread will update the flag
    if (particleIdx == 0) {
        convergeFlags[simulationIdx] = s_converged;
    }
    
    // Before we potentially overwrote in the same simulateKernel call, we can reduce global access this way
    pos[idx] = glm::vec4(x_new, 0.0f);
    vel[idx] = glm::vec4(v_new, 0.0f);
}

void launchSimulations(std::ostream& output_buf, glm::vec4* pos_buf, vector<float>& h_worldConvergenceTimes) {
    size_t maxBlocks(g_maxBlocks), numWorlds(g_numWorlds);
    int batch_ct = (numWorlds + maxBlocks - 1) / maxBlocks;

    auto t_batchLoopStart = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < batch_ct; i++) {
        int batch_offset = i * maxBlocks;
        int batch_sz = std::min(maxBlocks, numWorlds - batch_offset);

        // We should offset our pointers correspond to the correct batch
        glm::vec4* pos = g_particles.d_position + (batch_offset * g_numParticles);
        glm::vec4* vel = g_particles.d_velocity + (batch_offset * g_numParticles);
        float* radii = g_particles.d_radii + (batch_offset * g_numParticles);
        int* c_flags = g_particles.d_convergenceFlags + (batch_offset);

        // If specified, we will output a specific world's position data over time for each particle
        if ((g_curStep % 500 == 0) && g_worldLogIdx != -1 && g_worldLogIdx >= batch_offset && g_worldLogIdx < batch_offset + batch_sz) {
            int world_offset = (g_worldLogIdx - batch_offset) * g_numParticles;
            hipMemcpy(pos_buf, pos + world_offset, g_numParticles * sizeof(glm::vec4), hipMemcpyDeviceToHost);

            // csv format of |cur_step|cur_time|particle|x|y|z|
            for (int p = 0; p < g_numParticles; p++) {
                output_buf << g_curStep << "," << g_curStepTime << "," << p << "," << pos_buf[p].x << "," << pos_buf[p].y << "," << pos_buf[p].z << '\n';
            }
        }

        // Launch kernel, static size shared memory should be 64 * sizeof(glm::vec3) ~ 700 bytes per block should be ok
        // https://developer.nvidia.com/blog/using-shared-memory-cuda-cc/#static_shared_memory
        gpuErrchk(hipEventRecord(cudaEvt_simStart));
        simulateKernel<<<batch_sz, g_threadsPerBlock>>>(pos, vel, radii, c_flags);
        gpuErrchk(hipEventRecord(cudaEvt_simStop));
        gpuErrchk(hipEventSynchronize(cudaEvt_simStop));

        // FIXME: Do we need to sync here? 
        // gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipGetLastError());
    }
    auto t_batchLoopStop = std::chrono::high_resolution_clock::now();

    // Global Convergence //
    bool is_globalConverged = true;
    hipMemcpy(h_convergenceFlags, g_particles.d_convergenceFlags, numWorlds * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < numWorlds; i++) {
        is_globalConverged = is_globalConverged && h_convergenceFlags[i];
        
        if (BENCHMARK && h_convergenceFlags[i] && h_worldConvergenceTimes[i] < 0.0f) {
            auto conv_time = std::chrono::high_resolution_clock::now() - g_progStart;
            float conv_time_ms = std::chrono::duration<float, std::milli>(conv_time).count();
            
            h_worldConvergenceTimes[i] = conv_time_ms;
        }
    }

    // Could just set it equal, but this way we avoid global access :)
    if (is_globalConverged) {
        g_isGlobalConverged = true;
    }

    // Benchmarking //
    if (BENCHMARK) {
        float t_kernel;
        hipEventElapsedTime(&t_kernel, cudaEvt_simStart, cudaEvt_simStop);
        g_totalKernelTime += t_kernel;

        float t_batchLoopTime = std::chrono::duration<float, std::milli>(t_batchLoopStop - t_batchLoopStart).count();
        g_totalBatchLoopTime += t_batchLoopTime;

        g_timeSampleCt++;
    }
}

int main(int argc, char**argv) {
    if (argc < 2 || argc == 3) {
        cout << "Usage: ./executable <number of worlds/blocks> [world idx to log] [output file directory=../test/results/simdata] " << endl;
        return 0;
    }

    g_numWorlds = (size_t)std::stoull(argv[1]);
    if (g_numWorlds <= 0) {
        cerr << "Number of worlds must be > 0" << endl;
        return 1;
    }

    // Assuming world index AND output directory are given, then we will view 
    glm::vec4* pos_buf = nullptr;
    if (argc == 4) {
        g_worldLogIdx = std::stoi(argv[2]);
        g_worldLogOutDir = string(argv[3]);
        pos_buf = new glm::vec4[g_numParticles];

        if (g_worldLogIdx >= g_numWorlds) {
            cerr << "World log index must be in [0, numWorlds)!" << endl;
            return 1;
        }

        // If missing '/' don't exit, just add it
        if (g_worldLogOutDir[g_worldLogOutDir.size() - 1] != '/') {
            g_worldLogOutDir += "/";
        }

        cout << "[ALOG] Writing state output to " << g_worldLogOutDir << " for world " << g_worldLogIdx << endl;

        // https://stackoverflow.com/questions/16357999/current-date-and-time-as-string
        auto t = std::time(0);
        auto tm = *std::localtime(&t);

        std::ostringstream oss;
        oss << std::put_time(&tm, "%M_%S");

        string output_fname = "world_" + std::to_string(g_worldLogIdx) + "_" + oss.str() + ".csv";
        g_worldLogStream = std::ofstream(g_worldLogOutDir + output_fname);
        g_worldLogStream << "step,time,particle,x,y,z\n";
    }

    // Get GPU info https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#l2-cache-set-aside-for-persisting-accesses
    hipDeviceProp_t deviceProp;
    int device;

    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);

    printf("[INFO] Max grid sizes per dimension are x = %d, y = %d, z = %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf("[INFO] Max threads per block: %zu, max shared memory (b): %zu, L2 cache size (b): %zu, global memory size (b): %zu\n", deviceProp.maxThreadsPerBlock, deviceProp.sharedMemPerBlock, deviceProp.l2CacheSize, deviceProp.totalGlobalMem);
    // There will never be a case where we need more than 10 million blocks as we can only hipMalloc so much.
    g_maxBlocks = min((size_t)deviceProp.maxGridSize[0], (size_t)(1 << 28) - 1);
    printf("[INFO] Batching in %zu worlds / %zu max blocks\n", g_numWorlds, g_maxBlocks);
    
    g_threadsPerBlock = dim3(g_numParticles);
    printf("[INFO] Setting g_blocksPerGrid = dim3(min(%zu, %zu))\n", g_numWorlds, g_maxBlocks);
    g_blocksPerGrid = dim3(std::min(g_numWorlds, g_maxBlocks));

    // Initialize planes, particles, cuda buffers
    init();

    // Program converges when the last moving particle "stops", or the max time is exceeded.
    g_progStart = std::chrono::high_resolution_clock::now();
    auto end = g_progStart + std::chrono::seconds(MAX_SIMULATE_TIME_SECONDS);

    vector<float> h_worldConvergenceTimes(g_numWorlds, -1.0f);
    while (!g_isGlobalConverged && (std::chrono::high_resolution_clock::now() < end)) {
        launchSimulations(g_worldLogStream, pos_buf, h_worldConvergenceTimes);
        
        g_curStep++;
        g_curStepTime = g_curStep * DT_SIMULATION;
    }
    
    // Convergence time
    auto conv_time = std::chrono::high_resolution_clock::now() - g_progStart;
    auto conv_time_ms = std::chrono::duration_cast<std::chrono::milliseconds>(conv_time).count();
    printf("[BENCHMARK] Actual program time: %ld ms\n", conv_time_ms);

    // Print Timings //
    if (BENCHMARK) {
        float overall = g_totalKernelTime;
        float avg = g_totalKernelTime / g_timeSampleCt;
        float usage = g_totalKernelTime / (conv_time_ms);

        printf("[BENCHMARK] threadsPerBlock=particlesPerWorld: %d, blocksPerGrid=numWorlds: %d\n", g_threadsPerBlock.x, g_blocksPerGrid.x);
        printf("[BENCHMARK] Average individual simulateKernel() time over %d samples: %f ms\n", g_timeSampleCt, avg);
        printf("[BENCHMARK] Total time spent in kernel before global convergence: %f ms\n", overall);
        printf("[BENCHMARK] Kernel time / total program time: %f\n", usage);
        printf("[BENCHMARK] ----------------------------------\n");
        printf("[BENCHMARK] Total time sending and executing batches of simulateKernel(): %f ms\n", g_totalBatchLoopTime);
        printf("[BENCHMARK] Average time per batch loop: %f ms\n", g_totalBatchLoopTime / g_timeSampleCt);

        float minConvTime(std::numeric_limits<float>::max()), maxConvTime(0.0f), avgConvTime(0.0f);
        for (int i = 0; i < g_numWorlds; i++) {
            if (h_worldConvergenceTimes[i] < minConvTime) {
                minConvTime = h_worldConvergenceTimes[i];
            }

            if (h_worldConvergenceTimes[i] > maxConvTime) {
                maxConvTime = h_worldConvergenceTimes[i];
            }

            avgConvTime += h_worldConvergenceTimes[i];
        }
        
        avgConvTime /= g_numWorlds;
        printf("[BENCHMARK] (Local) Min convergence time: %f ms, Max convergence time: %f ms, Avg convergence time: %f ms\n", minConvTime, maxConvTime, avgConvTime);
    }

    // Cleanup //
    hipEventDestroy(cudaEvt_simStart);
    hipEventDestroy(cudaEvt_simStop);
    delete[] h_convergenceFlags;
    delete[] pos_buf;
    if (g_worldLogIdx != -1) {
        g_worldLogStream.close();
    }

    return 0;
}
