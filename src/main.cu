#include "hip/hip_runtime.h"
#include "include.h"

#include "constants.h"

#include <hip/hip_runtime.h>
#include <>

#include <chrono>

using std::cout, std::cerr, std::endl;
using std::vector, std::string, std::make_shared, std::shared_ptr;
using std::stoi, std::stoul, std::min, std::max, std::numeric_limits, std::abs;

/*
    3. Scaling / Output
        - Should be concise, don't need 1000 plots
        - We need to benchmark 1, 2, 4, 8, 16, 32, 64, 128 worlds
        - We should evaluate convergence time and maybe some other measure?
            i) Could do the min, avg, max of time convergence per worldcount
        - Can provide 1-2 insightful NCU charts and what they meant for optimization

    4. Optimization
        - Use NCU / timings to evaluate the bottlenecks, from there document
          rewrites
        - After code is improved as much as possible, can start plotting

*/

// FILE OUTPUT //
size_t g_worldLogIdx = -1;
string g_worldLogOutDir = "";
std::ofstream g_worldLogStream;

static const size_t g_numParticles = NUM_PARTICLES;
static size_t g_numWorlds, g_maxBlocks;
static int* h_convergenceFlags;
static bool g_isGlobalConverged(false);
float g_curTime(0.0f);
long long g_curStep(0);

// Device Hyperparameters - Constant Space //
__constant__ size_t d_numParticles;
__constant__ size_t d_numWorlds, d_numPlanes;
__constant__ glm::vec4 d_planeP[6], d_planeN[6];

bool g_is_simFrozen(false);
hipEvent_t kernel_simStart, kernel_simStop;

// blocks = k, threads = n
dim3 g_blocksPerGrid;
dim3 g_threadsPerBlock;

// static const int g_timeSampleSz = KERNEL_TIMING_SAMPLESZ;
static size_t g_timeSampleCt = 0;
static float g_totalKernelTimes = 0.0f;

ParticleData g_particles;
PlaneData g_planes;

static void init() {
    srand(0);
    size_t k = g_numWorlds;

    // CUDA //
        gpuErrchk(hipSetDevice(0));
        hipEventCreate(&kernel_simStart);
        hipEventCreate(&kernel_simStop);

    // Planes //
        const float plane_width = 540.0f;
        g_planes = PlaneData(6, plane_width);
        g_planes.initPlanes();
        g_planes.copyToDevice();

    // Particles //
        g_particles = ParticleData(g_numParticles, k);
        g_particles.init(0.5f);
        g_particles.copyToDevice();

        // numWorlds
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_numWorlds), &g_numWorlds, sizeof(size_t)));

        // We should zero h_convergenceFlags here and send that to CUDA
        h_convergenceFlags = new int[k];
        memset(h_convergenceFlags, 0, k * sizeof(int));
        hipMemcpy(g_particles.d_convergenceFlags, h_convergenceFlags, k * sizeof(int), hipMemcpyHostToDevice);
}

/*
    Instead of iterating over each particle, we will make a kernel that runs for each particle
*/

// Assume mass is 1; F / 1 = A
__device__ glm::vec3 getAcceleration(int idx, const glm::vec4* v) {
    float mass = 1.0f;

    // Simple force composed of gravity and air resistance
    glm::vec3 F_total = glm::vec3(0.0f, GRAVITY, 0.0f) - ((AIR_FRICTION / mass) * glm::vec3(v[idx])); 

    return F_total;
}

/*
    In a flocking simulation, you might have various rules and applications to follow - in this case, we are just
    going to establish a simple distance constraint that is resolve with impulse / momentum.
*/

static __device__ void solveConstraints(int idx, const glm::vec4* pos, const glm::vec4* vel, const float* radii,
                                 glm::vec3& x_new, glm::vec3& v_new, float& dt, const glm::vec3& a,
                                 int simulationIdx, int particleIdx, glm::vec3* s_dv) {
    // Truncate the w component
    const glm::vec3 x(pos[idx]), v(vel[idx]);
    const float r_i = radii[idx];

    // Particle-Particle Collisions //
    /*
        This could be below plane collisions, but seeing as we synchronized threads, we will do it here

        This is the inner loop of for i in range(particles), for j in range(i + 1, particles)

        We can grab the global array value as simulationIdx * particles + j. Note that because we
        are handling j > i particles in the ith thread, the jth thread will never see i - so we should
        update the opposite of the impulse from i -> j to the jth shared velocity; 
    */
    for (int j = particleIdx + 1; j < d_numParticles; j++) {
        int idx_j = simulationIdx * d_numParticles + j;

        glm::vec3 x_j(pos[idx_j]), v_j(vel[idx_j]);
        float r_j = radii[idx_j];

        /*
            If the distance from our particle to the other is less than radii[i] + radii[j] we have collided.

            We can take the direction of j to x and say we (particle i) should be pushed in that direction.

            The extent to which we move, or impulse, is dependent on the relative velocity, or how fast we
            are moving towards each other. For example, if v_rel < 0, we are moving towards each other, and we
            should push off more.

            J = [(1 + e) * v_rel] / [1/m1 + 1/m2]
        */
        glm::vec3 x_ij = x - x_j;
        float d_ij = glm::length(x_ij);

        if (d_ij < r_i + r_j) {
            glm::vec3 n_ij = glm::normalize(x_ij);
            glm::vec3 v_ij = v - v_j;

            float v_rel = glm::dot(v_ij, n_ij);
            float impulse = (1 + RESTITUTION) * v_rel / (1 + 1);

            glm::vec3 impulse_vec = impulse * n_ij;
            s_dv[particleIdx] += impulse_vec;

            // Consequently, we should change the velocities of j. This is not thread safe so we have to atomic
            atomicAdd(&s_dv[j].x, -impulse_vec.x);
            atomicAdd(&s_dv[j].y, -impulse_vec.y);
            atomicAdd(&s_dv[j].z, -impulse_vec.z);
        }
    }

    // Synchronize threads, because we don't want to start plane collisions until this is done
    __syncthreads();
    
    // Plane Collisions //
    for (int i = 0; i < d_numPlanes; i++) {
        const glm::vec3 p(d_planeP[i]), n(d_planeN[i]);

        glm::vec3 new_p = p + (r_i * n);

        float d_0 = glm::dot(x - new_p, n);
        float d_n = glm::dot(x_new - new_p, n);

        glm::vec3 v_tan = v - (glm::dot(v, n) * n);
        v_tan = (1 - FRICTION) * v_tan;

        if (d_n < FLOAT_EPS) {
            float f = d_0 / (d_0 - d_n);
            dt = f * dt;

            glm::vec3 v_collision = (v + (dt * a)) * RESTITUTION;    
            glm::vec3 x_collision = x;

            x_new = x_collision;
            v_new = (abs(glm::dot(v_collision, n)) * n) + (v_tan);
        }
    }
}

/*
    We are now working with global arrays of great size; 
*/
__global__ void simulateKernel(glm::vec4* pos, glm::vec4* vel, float* radii, int* convergeFlags) {
    unsigned int simulationIdx(blockIdx.x), particleIdx(threadIdx.x);

    // Overflow shouldn't be possible
    int idx = simulationIdx * d_numParticles + particleIdx;

    // FIXME: When a thread returns early, it cannot join __syncthreads later, so we should note this
    if (idx >= (d_numWorlds * d_numParticles)) {
        printf("Returning idx = %d\n", idx);        
        return;
    }

    // Allocate shared memory for graceful impulse & convergence handling; each block is one world so this works 
    __shared__ glm::vec3 s_dv[NUM_PARTICLES];
    __shared__ int s_converged;

    // We only want to initialize it once
    if (particleIdx == 0) {
        s_converged = 1;
    }

    // Handle fractional timesteps
    float dt_remaining = DT_SIMULATION;
    float dt = dt_remaining;
    short max_iter = 10;
    
    glm::vec3 x_cur(pos[idx]), v_cur(vel[idx]);
    glm::vec3 x_new(x_cur), v_new(v_cur);

    // printf("1\n");
    while (max_iter && dt_remaining > FLOAT_EPS) {
        // Within the timestep multiple collisions are possible, so we will have to reuse the shared memory 
        s_dv[particleIdx] = glm::vec3(0.0f);
        
        glm::vec3 a = getAcceleration(idx, vel);

        // Integrate over timestep to update
        x_new = x_cur + (dt * v_cur);
        v_new = v_cur + (dt * a);

        // We have to synchronize before and after entering
        __syncthreads();

        // Resolve particle-particle AND particle-plane position constraints
        solveConstraints(idx, pos, vel, radii, x_new, v_new, dt, a, simulationIdx, particleIdx, s_dv);

        __syncthreads();

        x_cur = x_new;
        v_cur = v_new;

        dt_remaining -= dt;
        max_iter--;
    }
    // printf("2\n");

    // We can do our convergence check here
    int is_stopped = 0;
    if (length(v_new) < STOP_VELOCITY) {
        v_new = glm::vec3(0.0f);
        is_stopped = 1;
    }

    // The and of all 64 particles being stopped in this world being 1 represents full convergence
    atomicAnd(&s_converged, is_stopped);

    // printf("3\n");
    // __syncthreads();
    // printf("4\n");

    // No need for atomic here, only the first thread will update the flag
    if (particleIdx == 0) {
        convergeFlags[simulationIdx] = s_converged;
    }
    
    // Before we potentially overwrote in the same simulateKernel call, we can reduce global access this way
    pos[idx] = glm::vec4(x_new, 0.0f);
    vel[idx] = glm::vec4(v_new, 0.0f);
}

void launchSimulations(std::ostream& output_buf, glm::vec4* pos_buf) {
    int maxBlocks(g_maxBlocks), numWorlds(g_numWorlds);
    int batch_ct = (numWorlds + maxBlocks - 1) / maxBlocks;

    gpuErrchk(hipEventRecord(kernel_simStart));

    for (int i = 0; i < batch_ct; i++) {
        int batch_offset = i * maxBlocks;
        int batch_sz = std::min(maxBlocks, numWorlds - batch_offset);

        // We should offset our pointers correspond to the correct batch
        glm::vec4* pos = g_particles.d_position + (batch_offset * g_numParticles);
        glm::vec4* vel = g_particles.d_velocity + (batch_offset * g_numParticles);
        float* radii = g_particles.d_radii + (batch_offset * g_numParticles);
        int* c_flags = g_particles.d_convergenceFlags + (batch_offset);

        // If specified, we will output a specific world's position data over time for each particle
        if ((g_curStep % 500 == 0) && g_worldLogIdx != -1 && g_worldLogIdx >= batch_offset && g_worldLogIdx < batch_offset + batch_sz) {
            int world_offset = (g_worldLogIdx - batch_offset) * g_numParticles;
            hipMemcpy(pos_buf, pos + world_offset, g_numParticles * sizeof(glm::vec4), hipMemcpyDeviceToHost);

            // csv format of |cur_step|cur_time|particle|x|y|z|
            for (int p = 0; p < g_numParticles; p++) {
                output_buf << g_curStep << "," << g_curTime << "," << p << "," << pos_buf[p].x << "," << pos_buf[p].y << "," << pos_buf[p].z << '\n';
            }
        }

        // Launch kernel, static size shared memory should be 64 * sizeof(glm::vec3) ~ 700 bytes per block should be ok
        // https://developer.nvidia.com/blog/using-shared-memory-cuda-cc/#static_shared_memory
        simulateKernel<<<batch_sz, g_threadsPerBlock>>>(pos, vel, radii, c_flags);

        // FIXME: Do we need to sync here? 
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipGetLastError());
    }

    // Test copy buffer
    // if (g_curStep % 500 == 0 && g_curStep > 0) {
    //     glm::vec4* pos_buffer = new glm::vec4[g_numParticles];
    //     hipMemcpy(pos_buffer, g_particles.d_position, g_numParticles * sizeof(glm::vec4), hipMemcpyDeviceToHost);
    //     for (int i = 0; i < 3; i++) {
    //         printf("Particle %d Y: %f\n", i, pos_buffer[i].y);
    //     }
    //     printf("------------\n");
    // }

    gpuErrchk(hipEventRecord(kernel_simStop));
    gpuErrchk(hipEventSynchronize(kernel_simStop));

    // Global Convergence //
    bool is_globalConverged = true;
    hipMemcpy(h_convergenceFlags, g_particles.d_convergenceFlags, numWorlds * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < numWorlds; i++) {
        is_globalConverged = is_globalConverged && h_convergenceFlags[i];
    }

    // Could just set it equal, but this way we avoid global access :)
    // I guess this begs the question: is time(local read + evaluate) < (global read + write) ?
    if (is_globalConverged) {
        g_isGlobalConverged = true;
    }

    // Benchmarking //
    if (BENCHMARK) {
        float kernel_time;
        hipEventElapsedTime(&kernel_time, kernel_simStart, kernel_simStop);
        g_totalKernelTimes += kernel_time;
        g_timeSampleCt++;
    }
}

int main(int argc, char**argv) {
    if (argc < 2 || argc == 4) {
        cout << "Usage: ./executable <number of worlds/blocks> [world idx to log] [output file directory] " << endl;
        return 0;
    }

    g_numWorlds = (size_t)std::stoull(argv[1]);
    if (g_numWorlds <= 0) {
        cerr << "Number of worlds must be > 0" << endl;
        return 1;
    }

    // Assuming world index AND output directory are given, then we will view 
    glm::vec4* pos_buf = nullptr;
    if (argc == 5) {
        g_worldLogIdx = (size_t)std::stoull(argv[2]);
        g_worldLogOutDir = string(argv[3]);
        pos_buf = new glm::vec4[g_numParticles];

        if (g_worldLogIdx >= g_numWorlds) {
            cerr << "World log index must be in [0, numWorlds)!" << endl;
            return 1;
        }

        // If missing '/' don't exit, just add it
        if (g_worldLogOutDir[g_worldLogOutDir.size() - 1] != '/') {
            g_worldLogOutDir += "/";
        }

        g_worldLogStream = std::ofstream(g_worldLogOutDir + "world_" + std::to_string(g_worldLogIdx) + ".csv");
        g_worldLogStream << "step,time,particle,x,y,z\n";
    }

    // Get GPU info https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#l2-cache-set-aside-for-persisting-accesses
    hipDeviceProp_t deviceProp;
    int device;

    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);

    printf("Max grid sizes per dimension are x = %d, y = %d, z = %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf("Max threads per block: %zu, max shared memory (bytes): %zu, L2 cache size (bytes): %zu, global memory size: %zu\n", deviceProp.maxThreadsPerBlock, deviceProp.sharedMemPerBlock, deviceProp.l2CacheSize, deviceProp.totalGlobalMem);
    // FIXME: For now we literally get overflow if we do this because worlds * maxBlocks is used to calculate the bound
    g_maxBlocks = min((size_t)deviceProp.maxGridSize[0], (size_t)(1 << 16) - 1);

    printf("Batching in %zu worlds / %zu max blocks\n", g_numWorlds, g_maxBlocks);
    
    g_threadsPerBlock = dim3(g_numParticles);

    printf("Setting g_blocksPerGrid = dim3(min(%zu, %zu))\n", g_numWorlds, g_maxBlocks);
    g_blocksPerGrid = dim3(std::min(g_numWorlds, g_maxBlocks));

    // Initialize planes, particles, cuda buffers
    init();

    // Program converges when the last moving particle "stops", or the max time is exceeded.
    auto start = std::chrono::high_resolution_clock::now();
    auto end = start + std::chrono::seconds(MAX_SIMULATE_TIME_SECONDS);
    
    while (!g_isGlobalConverged && (std::chrono::high_resolution_clock::now() < end)) {
        launchSimulations(g_worldLogStream, pos_buf);
        
        g_curStep++;
        g_curTime = g_curStep * DT_SIMULATION;
    }
    
    // Convergence time
    auto conv_time = std::chrono::high_resolution_clock::now() - start;
    auto conv_time_ms = std::chrono::duration_cast<std::chrono::milliseconds>(conv_time).count();
    printf("Actual program time: %ld ms\n", conv_time_ms);

    // Print Timings //
    if (BENCHMARK) {
        float overall = g_totalKernelTimes;
        float avg = g_totalKernelTimes / g_timeSampleCt;
        float usage = g_totalKernelTimes / (conv_time_ms);

        printf("Number of threads: %d, number of blocks (per grid): %d\n", g_threadsPerBlock.x, g_blocksPerGrid.x);
        printf("Average simulateKernel() execution time over %d samples: %f ms\n", g_timeSampleCt, avg);
        printf("Overall kernel time before convergence: %f ms\n", overall);
        printf("Kernel time / total program time: %f\n", usage);
    }

    // Cleanup //
    hipEventDestroy(kernel_simStart);
    hipEventDestroy(kernel_simStop);
    delete[] h_convergenceFlags;
    if (g_worldLogIdx != -1) {
        g_worldLogStream.close();
        delete[] pos_buf;
    }

    return 0;
}
